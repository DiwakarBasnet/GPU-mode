
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define OUT_TILE_DIM 16
#define IN_TILE_DIM 8

#define c0 0.02f
#define c1 0.52f
#define c2 0.25f
#define c3 0.12f
#define c4 0.89f
#define c5 0.37f
#define c6 0.93f

__global__ void stencil_kernel(float *in, float *out, unsigned int N) {
    int iStart = blockIdx.z * OUT_TILE_DIM;
    int j = blockIdx.y * OUT_TILE_DIM + threadIdx.y - 1;
    int k = blockIdx.x * OUT_TILE_DIM + threadIdx.x - 1;

    __shared__ float inPrev_s[IN_TILE_DIM][IN_TILE_DIM];
    __shared__ float inCurr_s[IN_TILE_DIM][IN_TILE_DIM];
    __shared__ float inNext_s[IN_TILE_DIM][IN_TILE_DIM];

    if (iStart-1 >= 0 && iStart-1 < N && j >= 0 && j < N && k >= 0 && k < N) {
        inPrev_s[threadIdx.y][threadIdx.x] = in[(iStart - 1)*N*N + j*N + k];
    }
    if (iStart >= 0 && iStart < N && j >= 0 && j < N && k >= 0 && k < N) {
        inCurr_s[threadIdx.y][threadIdx.x] = in[iStart*N*N + j*N + k];
    }
    for (int i = iStart; i < iStart + OUT_TILE_DIM; i++) {
        if (i + 1 >= 0 && i + 1 < N && j >= 0 && j < N && k >= 0 && k < N) {
            inNext_s[threadIdx.y][threadIdx.x] = in[(i + 1)*N*N + j*N + k];
        }
        __syncthreads();
        if (i >= 1 && i < N - 1 && j >= 1 && j < N - 1 && k >= 1 && k < N - 1) {
            if (threadIdx.y >= 1 && threadIdx.y < IN_TILE_DIM - 1
                && threadIdx.x >= 1 && threadIdx.x < IN_TILE_DIM - 1) {
                    out[i*N*N + j*N + k] = c0*inCurr_s[threadIdx.y][threadIdx.x]
                                         + c1*inCurr_s[threadIdx.y][threadIdx.x-1]
                                         + c2*inCurr_s[threadIdx.y][threadIdx.x+1]
                                         + c3*inCurr_s[threadIdx.y-1][threadIdx.x]
                                         + c4*inCurr_s[threadIdx.y+1][threadIdx.x]
                                         + c5*inPrev_s[threadIdx.y][threadIdx.x]
                                         + c6*inNext_s[threadIdx.y][threadIdx.x];
                }
        }
        __syncthreads();
        inPrev_s[threadIdx.y][threadIdx.x] = inCurr_s[threadIdx.y][threadIdx.x];
        inCurr_s[threadIdx.y][threadIdx.x] = inNext_s[threadIdx.y][threadIdx.x];
    }
}

void stencil_sweep(float *in_h, float *out_h, unsigned int N) {
    int size = N * N * N * sizeof(float);

    float *in_d, *out_d;

    // Device memory allocation
    hipError_t err1 = hipMalloc((void**)&in_d, size);
    if (err1 != hipSuccess) {
        printf("%s in %s at line %d", hipGetErrorString(err1), __FILE__, __LINE__);
    }
    hipError_t err2 = hipMalloc((void**)&out_d, size);
    if (err2 != hipSuccess) {
        printf("%s in %s at line %d", hipGetErrorString(err2), __FILE__, __LINE__);
    }

    hipMemcpy(in_d, in_h, size, hipMemcpyHostToDevice);

    // Launch stencil kernel
    dim3 dimGrid((N + IN_TILE_DIM - 1)/IN_TILE_DIM, (N + IN_TILE_DIM - 1)/IN_TILE_DIM, (N + IN_TILE_DIM - 1)/IN_TILE_DIM);
    dim3 dimBlock(IN_TILE_DIM, IN_TILE_DIM, IN_TILE_DIM);

    stencil_kernel<<<dimGrid, dimBlock>>>(in_d, out_d, N);

    // Check kernel launch for errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("%s while kernel launch", hipGetErrorString(err));
    }

    // Copy kernel results from device to host
    hipMemcpy(out_h, out_d, size, hipMemcpyDeviceToHost);

    hipFree(in_d);
    hipFree(out_d);
}

void printStencil(float *stencil, int N) {
    for (int z = 0; z < N; z++) {
        for (int y = 0; y < N; y++) {
            for (int x = 0; x < N; x++) {
                printf("%.2f ", stencil[z*N*N + y*N + x]);
            }
            printf("\n");
        }
        printf("###############################################");
    }
}

int main() {
    int N = 16;
    int size = N * N * N * sizeof(float);

    float *in_h = (float *)malloc(size);
    float *out_h = (float *)malloc(size);

    for (int i = 0; i < N*N*N; i++) {
        in_h[i] = (float)(rand() % 10);
    }

    printf("\nOriginal Stencil:\n");
    printStencil(in_h, N);

    stencil_sweep(in_h, out_h, N);

    printf("\nResult Stencil:\n");
    printStencil(out_h, N);

    free(in_h);
    free(out_h);

    return 0;
}