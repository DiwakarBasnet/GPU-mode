#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define NUM_BINS 7

__global__ void histo_private_kernel(char *data, unsigned int length, unsigned int *histo) {
    // Initialilze privatized bins
    __shared__ unsigned int histo_s[NUM_BINS];
    for (unsigned int bin = threadIdx.x; bin < NUM_BINS; bin += blockDim.x) {
        histo_s[bin] = 0u;
    }
    __syncthreads();
    // Histogram
    unsigned int accumulator = 0;
    int prevBinIdx = -1;
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    for (unsigned int i = tid; i < length; i += blockDim.x * gridDim.x) {
        int alphabet_position = data[i] - 'a';
        if (alphabet_position >= 0 && alphabet_position < 26) {
            int bin = alphabet_position / 4;
            if (bin == prevBinIdx) {
                ++accumulator;
            } else {
                if (accumulator > 0) {
                    atomicAdd(&(histo_s[prevBinIdx]), accumulator);
                }
                accumulator = 1;
                prevBinIdx = bin;
            }
        }
    }
    if (accumulator > 0) {
        atomicAdd(&(histo_s[prevBinIdx]), accumulator);
    }
    __syncthreads();
    // Commit to global memory
    for (unsigned int bin = threadIdx.x; bin < NUM_BINS; bin += blockDim.x) {
        unsigned int binValue = histo_s[bin];
        if (binValue > 0) {
            atomicAdd(&(histo[bin]), binValue);
        }
    }
}

void histo_private(char *data_h, int length, unsigned int *histo_h) {
    int size_data = length * sizeof(char);
    int size_histo = NUM_BINS * sizeof(unsigned int);

    char *data_d;
    unsigned int *histo_d;

    // Allocate device memory
    hipError_t err1 = hipMalloc((void**)&data_d, size_data);
    if (err1 != hipSuccess) {
        printf("%s in %s at line %d", hipGetErrorString(err1), __FILE__, __LINE__);
    }
    hipError_t err2 = hipMalloc((void**)&histo_d, size_histo);
    if (err2 != hipSuccess) {
        printf("%s in %s at line %d", hipGetErrorString(err2), __FILE__, __LINE__);
    }

    hipMemcpy(data_d, data_h, size_data, hipMemcpyHostToDevice);

    // Kernel launch
    dim3 dimBlock(8);
    dim3 dimGrid((length + dimBlock.x - 1)/dimBlock.x);

    histo_private_kernel<<<dimGrid, dimBlock>>>(data_d, length, histo_d);

    // Check for kernel launch errors
    hipError_t err3 = hipGetLastError();
    if (err3 != hipSuccess) {
        printf("%s in %s at line %d", hipGetErrorString(err3), __FILE__, __LINE__);
    }

    // Copy results from device to host
    hipMemcpy(histo_h, histo_d, size_histo, hipMemcpyDeviceToHost);

    hipFree(histo_d);
    hipFree(data_d);
}

int main() {
    // Example input string (should be lowercase letters for this example)
    char data[] = "this is an example of a cuda histogram computation";
    unsigned int length = strlen(data);

    unsigned int histo_host[7] = {0};

    // Compute histogram on the GPU
    histo_private(data, length, histo_host);

    // Print the histogram
    for (int i = 0; i < 7; i++) {
        printf("b%d: %u\n", i, histo_host[i]);
    }

    return 0;
}