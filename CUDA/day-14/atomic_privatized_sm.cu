#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define NUM_BINS 7

__global__ void histo_private_kernel(char *data, unsigned int length, unsigned int *histo) {
    // Initialize privatized bins
    __shared__ unsigned int histo_s[NUM_BINS];
    for (unsigned int bin = threadIdx.x; bin < NUM_BINS; bin += blockDim.x) {
        histo_s[bin] = 0u;
    }
    __syncthreads();
    // Histogram
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < length) {
        int alphabet_position = data[i] - 'a';
        if (alphabet_position >= 0 && alphabet_position < 26) {
            atomicAdd(&(histo_s[alphabet_position/4]), 1);
        }
    }
    __syncthreads();
    // Commit to global memory
    for (unsigned int bin = threadIdx.x; bin < NUM_BINS; bin += blockDim.x) {
        unsigned int binValue = histo_s[bin];
        if (binValue > 0) {
            atomicAdd(&(histo[bin]), binValue);
        }
    }
}

void histo_private(char *data_h, unsigned int length, unsigned int *histo_h) {
    int size_data = length * sizeof(char);
    int size_histo = 7 * sizeof(unsigned int);

    char *data_d;
    unsigned int *histo_d;

    // Device memory allocation
    hipError_t err1 = hipMalloc((void**)&data_d, size_data);
    if (err1 != hipSuccess) {
        printf("%s in %s at line %d", hipGetErrorString(err1), __FILE__, __LINE__);
    }
    hipError_t err2 = hipMalloc((void**)&histo_d, size_histo);
    if (err2 != hipSuccess) {
        printf("%s in %s at line %d", hipGetErrorString(err2), __FILE__, __LINE__);
    }

    hipMemcpy(data_d, data_h, size_data, hipMemcpyHostToDevice);

    // Initialize device histo with 0
    hipError_t err3 = hipMemset(histo_d, 0, size_histo);
    if (err3 != hipSuccess) {
        printf("%s in %s at line %d", hipGetErrorString(err3), __FILE__, __LINE__);
    }

    // Kernel Initialization
    dim3 dimBlock(32);
    dim3 dimGrid((length + dimBlock.x - 1)/dimBlock.x);

    histo_private_kernel<<<dimGrid, dimBlock>>>(data_d, length, histo_d);

    // Copy result from device to host
    hipMemcpy(histo_h, histo_d, size_histo, hipMemcpyDeviceToHost);

    hipFree(histo_d);
    hipFree(data_d);
}

int main() {
    // Example input string (should be lowercase letters for this example)
    char data[] = "this is an example of a cuda histogram computation";
    unsigned int length = strlen(data);

    unsigned int histo_host[7] = {0};

    // Compute histogram on the GPU
    histo_private(data, length, histo_host);

    // Print the histogram
    for (int i = 0; i < 7; i++) {
        printf("b%d: %u\n", i, histo_host[i]);
    }

    return 0;
}