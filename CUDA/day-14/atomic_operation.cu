#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>

// Kernel computes histogram for letters 'a' to 'z'
__global__ void histo_kernel(const char *data, unsigned int length, unsigned int *histo) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < length) {
        int alphabet_position = data[i] - 'a';
        if (alphabet_position >= 0 && alphabet_position < 26) {
            // Each thread atomically adds to the corresponding bin
            atomicAdd(&histo[alphabet_position/4], 1);
        }
    }
}

void histo(char *data_h, unsigned int length, unsigned int *histo_h) {
    int data_size = length * sizeof(char);
    int histo_size = 7 * sizeof(unsigned int);
    char *data_d;
    unsigned int *histo_d;

    // Allocate device memory
    hipError_t err1 = hipMalloc((void**)&data_d, data_size);
    if (err1 != hipSuccess) {
        printf("%s in %s at %d\n", hipGetErrorString(err1), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }

    hipError_t err2 = hipMalloc((void**)&histo_d, histo_size);
    if (err2 != hipSuccess) {
        printf("%s in %s at %d\n", hipGetErrorString(err2), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }

    // Initialize device histogram to zero
    hipError_t err3 = hipMemset(histo_d, 0, histo_size);
    if (err3 != hipSuccess) {
        printf("%s in %s at %d\n", hipGetErrorString(err3), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }

    // Copy input data from host to device
    hipMemcpy(data_d, data_h, data_size, hipMemcpyHostToDevice);

    // Kernel execution configuration
    dim3 dimBlock(32);
    dim3 dimGrid((length + dimBlock.x - 1) / dimBlock.x);

    histo_kernel<<<dimGrid, dimBlock>>>(data_d, length, histo_d);

    // Check for errors after kernel launch
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("%s in %s at %d\n", hipGetErrorString(err), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }

    hipDeviceSynchronize();

    // Copy the histogram from device to host
    hipMemcpy(histo_h, histo_d, histo_size, hipMemcpyDeviceToHost);

    hipFree(data_d);
    hipFree(histo_d);
}

int main() {
    // Example input string (should be lowercase letters for this example)
    char data[] = "this is an example of a cuda histogram computation";
    unsigned int length = strlen(data);

    unsigned int histo_host[7] = {0};

    // Compute histogram on the GPU
    histo(data, length, histo_host);

    // Print the histogram
    for (int i = 0; i < 7; i++) {
        printf("b%d: %u\n", i, histo_host[i]);
    }

    return 0;
}
