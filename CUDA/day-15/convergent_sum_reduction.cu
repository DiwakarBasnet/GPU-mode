#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void ConvergentSumReductionKernel(float *input, float *output) {
    unsigned int i = threadIdx.x;
    for (unsigned int stride = blockDim.x; stride >= 1; stride /= 2) {
        if (threadIdx.x < stride) {
            input[i] += input[i + stride];
        }
        __syncthreads();
    }
    if (threadIdx.x == 0) {
        *output = input[0];
    }
}

void ConvergentSumReduction(float *input_h, float *output_h, int N) {
    int size = N * sizeof(float);
    float *input_d, *output_d;

    // Allocate device memory
    hipError_t err1 = hipMalloc((void**)&input_d, size);
    if (err1 != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err1), __FILE__, __LINE__);
    }
    hipError_t err2 = hipMalloc((void**)&output_d, sizeof(float));
    if (err2 != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err2), __FILE__, __LINE__);
    }

    hipMemcpy(input_d, input_h, size, hipMemcpyHostToDevice);

    // Launch kernel
    dim3 dimBlock(N / 2);
    dim3 dimGrid(1);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    ConvergentSumReductionKernel<<<dimGrid, dimBlock>>>(input_d, output_d);

    hipDeviceSynchronize();
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Time taken: %f ms\n", milliseconds);

    // Check for kernel launch errors
    hipError_t err3 = hipGetLastError();
    if (err3 != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err3), __FILE__, __LINE__);
    }

    // Copy result back to host
    hipMemcpy(output_h, output_d, sizeof(float), hipMemcpyDeviceToHost);

    hipFree(input_d);
    hipFree(output_d);
}

int main() {
    int N = 128;
    float *input_h = (float *)malloc(N * sizeof(float));
    float *output_h = (float *)malloc(sizeof(float));

    for (int i = 0; i < N; i++) {
        input_h[i] = i + 1;
    }

    ConvergentSumReduction(input_h, output_h, N);

    printf("\nSum: %f\n", *output_h);

    free(input_h);
    free(output_h);

    return 0;
}