#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void SimpleSumReductionKernel(float *input, float *output) {
    unsigned int i = 2*threadIdx.x;
    for (unsigned int stride = 1; stride <= blockDim.x; stride *= 2) {
        if (threadIdx.x % stride == 0) {
            input[i] += input[i + stride];
        }
        __syncthreads();
    }
    if (threadIdx.x == 0) {
        *output = input[0];
    }
}


void SimpleSumReduction(float *input_h, float *output_h, int N) {
    float *input_d, *output_d;
    int size = N * sizeof(float);

    // Allocate device memory
    hipError_t err1 = hipMalloc((void**)&input_d, size);
    if (err1 != hipSuccess) {
        printf("%s in %s at line %d", hipGetErrorString(err1), __FILE__, __LINE__);
    }
    hipError_t err2 = hipMalloc((void**)&output_d, sizeof(float));
    if (err2 != hipSuccess) {
        printf("%s in %s at line %d", hipGetErrorString(err2), __FILE__, __LINE__);
    }

    hipMemcpy(input_d, input_h, size, hipMemcpyHostToDevice);

    // Launch kernel
    dim3 dimBlock(N / 2);
    dim3 dimGrid(1);

    SimpleSumReductionKernel<<<dimGrid, dimBlock>>>(input_d, output_d);

    // Check for errors
    hipError_t err3 = hipGetLastError();
    if (err3 != hipSuccess) {
        printf("%s in %s at line %d", hipGetErrorString(err3), __FILE__, __LINE__);
    }

    // Copy result back to host from device
    hipMemcpy(output_h, output_d, sizeof(float), hipMemcpyDeviceToHost);

    hipFree(input_d);
    hipFree(output_d);
}

int main() {
    int N = 20;
    int size = N * sizeof(float);

    float *input_h = (float *)malloc(size);
    float *output_h = (float *)malloc(sizeof(float));

    for (int i = 0; i < N; i ++) {
        input_h[i] = i + 1;
    }

    SimpleSumReduction(input_h, output_h, N);

    printf("\nSum: %f\n", *output_h);

    free(input_h);
    free(output_h);

    return 0;
}