#include "hip/hip_runtime.h"
#define TILE_WIDTH 16
__global__ void matrixMulKernel(float *M, float *N, float *P, int Width){

    __shared__ float Mds[TILE_WIDTH][TILE_WIDTH];   // shared mem arrays
    __shared__ float Nds[TILE_WIDTH][TILE_WIDTH];   // scope of sma is blocks so 1 version of Mds and Nds will be created for each block

    int bx = blockIdx.x; int by = blockIdx.y;   // 1 version of bx, by, tx and ty will be created for each thread and will reside in registers that are accessible by the thread
    int tx = threadIdx.x; int ty = threadIdx.y; // Once thread ends, the values of these variables cease to exist

    // Identify the row and column of the P element to work on
    int Row = by * TILE_WIDTH + ty;
    int Col = bx * TILE_WIDTH + tx;

    // Loop over the M and N tiles required to compute P element
    float Pvalue = 0;   // Strip-mining (break long loops into phases)
    for (int ph = 0; ph < ceil(Width/(float)TILE_WIDTH); ++ph) {     // Each iteration corresponds to one phase of calculation

        // Collaborative loading of M and N tiles into shared memory
        if ((Row < Width) && (ph*TILE_WIDTH + tx) < Width) {
            Mds[ty][tx] = M[Row*Width + ph*TILE_WIDTH + tx];    // each phase uses one tile of M and one tile of N elements
        }
        else Mds[ty][tx] = 0.0f;
        if ((ph*TILE_WIDTH + ty) < Width && Col < Width) {
            Nds[ty][tx] = N[(ph*TILE_WIDTH + ty)*Width + Col];
        }
        else Nds[ty][tx] = 0.0f;
        __syncthreads();

        for (int k = 0; k < TILE_WIDTH; ++k) {
            Pvalue += Mds[ty][k] * Nds[k][tx];
        }
        __syncthreads();

    }
    if (Row < Width) && (Col < Width) {
        P[Row * Width + Col] = Pvalue;
    }

}