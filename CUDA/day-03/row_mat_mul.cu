
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

void printMatrix(float *matrix, int width, int height) {
    for (int i = 0; i < height; i++) {
        for (int j = 0; j < width; j++) {
            printf("%f ", matrix[i * width + j]);
        }
        printf("\n");
    }
}

__global__ void matrixMul_kernel(float *A, float *B, float *C, int N) {
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;

    if (thread_id < N) {
        for (int col = 0; col < N; col++) {
            // According to the question, we want each thread to produce
            // a row of the output matrix, which means that each thread/row
            // will loop through all the columns of the product matrix. 
            float c_sum = 0.0f;

            for (int k = 0; k < N; k++) {
                c_sum += A[thread_id * N + k] * B[k * N + col];
            }
            C[thread_id * N + col] = c_sum;
        }
    }
}

void matrixMul(float *A, float *B, float *C, int N) {
    int size = N * N * sizeof(float);
    float *A_d, *B_d, *C_d;

    // Part 1: Allocate device memory for A, B, and C.
    // copy A and B to device memory. 
    hipError_t err = hipMalloc((void**)&A_d, size);
    if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
    }
    err = hipMalloc((void**)&B_d, size);
    if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
    }
    err = hipMalloc((void**)&C_d, size);
    if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
    }
    
    hipMemcpy(A_d, A, size, hipMemcpyHostToDevice);
    hipMemcpy(B_d, B, size, hipMemcpyHostToDevice);

    // Part 2: Call the kernel to launch the grid of threads. 
    // to perform the matrix multiplication. 
    dim3 dimGrid(ceil(N / 32.0), 1, 1);
    dim3 dimBlock(32, 1, 1);
    matrixMul_kernel<<<dimGrid, dimBlock>>>(A_d, B_d, C_d, N);

    // Part 3: Copy the result back to the host. 
    // free the device memory. 
    hipMemcpy(C, C_d, size, hipMemcpyDeviceToHost);

    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
}

int main() {
    int N = 5;
    
    int size = N * N * sizeof(float);

    float *A = (float *)malloc(size);
    float *B = (float *)malloc(size);
    float *C = (float *)malloc(size);

    // Initialize the matrices. 
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            int offset = i * N + j;
            A[offset] = rand() % 6;
            B[offset] = rand() % 6;
        }
    }

    // Print the matrices. 
    printMatrix(A, N, N);
    printMatrix(B, N, N);

    // Call the matrix multiplication function. 
    matrixMul(A, B, C, N);

    // Print the result. 
    printMatrix(C, N, N);
    
    return 0;
}
