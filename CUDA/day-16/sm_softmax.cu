#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define BLOCK_DIM 8

__global__ void SoftmaxKernel(float *input, float *output, int width, int height) {
	extern __shared__ float smem[];

	int row = blockIdx.x;
	int tid = threadIdx.x;

	if (row >= height) return;

	float *input_row = input + row * width;
	float *output_row = output + row * width;
	float row_max, row_norm;

	if (tid == 0) {
			 row_max = -INFINITY;
			 row_norm = 0.0f;

			 for (int col = 0; col < width; col++) {
						float x = input_row[col];
						if (x > row_max) {
								 row_norm *= expf(row_max - x);
								 row_max = x;
						}
						row_norm += expf(x - row_max);
			 }
			 smem[0] = row_max;
			 smem[1] = row_norm;
	}
	__syncthreads();

	row_max = smem[0];
	row_norm = smem[1];

	for (int i = tid; i < width; i += blockDim.x) {
		output_row[i] = expf(input_row[i] - row_max)/row_norm;
	}
}

void Softmax(float *input_h, float *output_h, int width, int height) {
	int size = width * height * sizeof(float);
	float *input_d, *output_d;

	// Allocate device memory
	hipError_t err1 = hipMalloc((void**)&input_d, size);
	if (err1 != hipSuccess) {
		printf("%s in %s at line %d", hipGetErrorString(err1), __FILE__, __LINE__);
	}
	hipError_t err2 = hipMalloc((void**)&output_d, size);
	if (err2 != hipSuccess) {
		printf("%s in %s at line %d", hipGetErrorString(err2), __FILE__, __LINE__);
	}

	hipMemcpy(input_d, input_h, size, hipMemcpyHostToDevice);

	// Initialize kernel
	dim3 dimBlock(BLOCK_DIM);
	dim3 dimGrid(height);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	float ms = 0.0f;
	hipEventRecord(start);

	SoftmaxKernel<<<dimGrid, dimBlock, BLOCK_DIM * sizeof(float)>>>(input_d, output_d, width, height);

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&ms, start, stop);

	printf(">> Kernel execution time: %f ms\n", ms);

	// Check for kernel launch errors
	hipError_t err3 = hipGetLastError();
	if (err3 != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err3), __FILE__, __LINE__);
	}

	// Copy result from device to host
	hipMemcpy(output_h, output_d, size, hipMemcpyDeviceToHost);

	hipFree(input_d);
	hipFree(output_d);
}

void printMatrix(float *matrix, int height, int width) {
		for (int r = 0; r < height; r++) {
				 for (int c = 0; c < width; c++) {
							printf("%f ", matrix[r * width + c]);
				 }
				 printf("\n");
		}
}

int main() {
    int height = 8;
    int width = 8;
    int size = height * width * sizeof(float);

    float *input_h = (float *)malloc(size);
    float *output_h = (float *)malloc(size);

    for (int i = 0; i < height * width; i ++) {
        input_h[i] = (float)(rand() % 10);
    }

    printf("Original input:\n");
    printMatrix(input_h, height, width);

    Softmax(input_h, output_h, height, width);

    printf("\nSoftmax output\n");
    printMatrix(output_h, height, width);

    free(input_h);
    free(output_h);

    return 0;
}