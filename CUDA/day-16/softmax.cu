#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void SimpleSoftmaxKernel(float *input, float *output, int N) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= N) return;

    // Calculate exponentials for all elements
    output[idx] = expf(input[idx]);

    // Wait for all threads to finish exponentiation
    __syncthreads();

    // Compute sum of exponents
    float sum = 0.0f;
    for (int i = 0; i < N; i++) {
        sum += output[i];
    }

    // Normalize Values
    output[idx] = output[idx] / sum;

}


void SimpleSoftmax(float *input_h, float *output_h, int N) {
    float *input_d, *output_d;
    int size = N * sizeof(float);

    // Allocate device memory
    hipError_t err1 = hipMalloc((void**)&input_d, size);
    if (err1 != hipSuccess) {
        printf("%s in %s at line %d", hipGetErrorString(err1), __FILE__, __LINE__);
    }
    hipError_t err2 = hipMalloc((void**)&output_d, size);
    if (err2 != hipSuccess) {
        printf("%s in %s at line %d", hipGetErrorString(err2), __FILE__, __LINE__);
    }

    hipMemcpy(input_d, input_h, size, hipMemcpyHostToDevice);

    // Launch kernel
    dim3 dimBlock(N);
    dim3 dimGrid(1);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    SimpleSoftmaxKernel<<<dimGrid, dimBlock>>>(input_d, output_d, N);

    hipDeviceSynchronize();
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("\nTime taken: %f ms\n", milliseconds);

    // Check for errors
    hipError_t err3 = hipGetLastError();
    if (err3 != hipSuccess) {
        printf("%s in %s at line %d", hipGetErrorString(err3), __FILE__, __LINE__);
    }

    // Copy result back to host from device
    hipMemcpy(output_h, output_d, size, hipMemcpyDeviceToHost);

    hipFree(input_d);
    hipFree(output_d);
}

int main() {
    int N = 8;
    int size = N * sizeof(float);

    float *input_h = (float *)malloc(size);
    float *output_h = (float *)malloc(size);

    for (int i = 0; i < N; i ++) {
        input_h[i] = (float)(rand() % 10);
    }

    printf("Original input:\n");
    for (int i = 0; i < N; i++) {
        printf("%f ", input_h[i]);
    }

    SimpleSoftmax(input_h, output_h, N);

    printf("\nSoftmax output\n");
    for (int i = 0; i < N; i++) {
        printf("%f ", output_h[i]);
    }

    free(input_h);
    free(output_h);

    return 0;
}