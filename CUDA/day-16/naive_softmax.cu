#include "hip/hip_runtime.h"
%%cuda
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void NaiveSoftmaxKernel(float *input, float *output, int height, int width) {
    
    int row = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < height) {
        // max of this row
        float x_max = -INFINITY;
        // norm factor of this row
        float norm = 0.0f;

        // Pass 1: Calculation fo max
        for (int col = 0; col < width; col++) {
            int i = row * width + col;
            x_max = max(x_max, input[i]);
        }
        // Pass 2: Calculation fo normalized sum
        for (int col = 0; col < width; col++) {
            int i = row * width + col;
            norm += expf(input[i] - x_max);
        }
        // Pass 3: Softmax calculation
        for (int col = 0; col < width; col++) {
            int i = row * width + col;
            output[i] = expf(input[i] - x_max) / norm;
        }
    }
}


void NaiveSoftmax(float *input_h, float *output_h, int height, int width) {
    float *input_d, *output_d;
    int size = height * width * sizeof(float);

    // Allocate device memory
    hipError_t err1 = hipMalloc((void**)&input_d, size);
    if (err1 != hipSuccess) {
        printf("%s in %s at line %d", hipGetErrorString(err1), __FILE__, __LINE__);
    }
    hipError_t err2 = hipMalloc((void**)&output_d, size);
    if (err2 != hipSuccess) {
        printf("%s in %s at line %d", hipGetErrorString(err2), __FILE__, __LINE__);
    }

    hipMemcpy(input_d, input_h, size, hipMemcpyHostToDevice);

    // Launch kernel
    dim3 dimBlock(8);
    dim3 dimGrid((height + dimBlock.x - 1)/dimBlock.x);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    NaiveSoftmaxKernel<<<dimGrid, dimBlock>>>(input_d, output_d, height, width);

    hipDeviceSynchronize();
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("\nTime taken: %f ms\n", milliseconds);

    // Check for errors
    hipError_t err3 = hipGetLastError();
    if (err3 != hipSuccess) {
        printf("%s in %s at line %d", hipGetErrorString(err3), __FILE__, __LINE__);
    }

    // Copy result back to host from device
    hipMemcpy(output_h, output_d, size, hipMemcpyDeviceToHost);

    hipFree(input_d);
    hipFree(output_d);
}

void printMatrix(float *matrix, int height, int width) {
  for (int r = 0; r < height; r++) {
      for (int c = 0; c < width; c++) {
          printf("%f ", matrix[r * width + c]);
      }
      printf("\n");
  }
}

int main() {
    int height = 8;
    int width = 8;
    int size = height * width * sizeof(float);

    float *input_h = (float *)malloc(size);
    float *output_h = (float *)malloc(size);

    for (int i = 0; i < height * width; i ++) {
        input_h[i] = (float)(rand() % 10);
    }

    printf("Original input:\n");
    printMatrix(input_h, height, width);

    NaiveSoftmax(input_h, output_h, height, width);

    printf("\nSoftmax output\n");
    printMatrix(output_h, height, width);

    free(input_h);
    free(output_h);

    return 0;
}
